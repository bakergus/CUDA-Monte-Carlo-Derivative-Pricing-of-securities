#include "hip/hip_runtime.h"
project
=======
#define MAX(a,b) ((a) > (b) ? a : b)
#define MIN(a,b) ((a) < (b) ? a : b)

__global__ void sum_simple_kernel(float *X, float Y, float *max, float *min, float maximum, float minimum)
{
__shared__ float partialSum[];
unsigned int t=threadIdx.x;
for(unsigned int stride=1;stride<blockDim.x;stride*=2)
{
__syncthreads();
if(t % (2*stride) ==0)
partialSum[t]+=partialSum[t+stride];
max[t]= Max(X[t],X[t+stride]);
min[t]= Min(X[t],X[t+stride]);
}
Y=partialSum[blockDim.x];
maximum = max[blockDim.x];
minimum = min[blockDim.x];
}

