#include "hip/hip_runtime.h"
/*------------------------------------------------------
         Template taken from vecadd
--------------------------------------------------------*/

#include <stdio.h>
#include <sys/time.h>
#include "kernel.cu"

#define M 20 //number of assets
#define N 200 //number of simus
#define blocksize 32


static void launch_simple_kernel(float *X, float Y, float *max, float *min, float maximum, float minimum)
 {
    dim3 dimGrid(blocksize,1,1);
    dim3 dimBlock(M,(N-1)/blocksize+1,1);
    sum_simple_kernel<<<dimGrid,dimBlock>>>(float *X, float Y, float *max, float *min, float maximum, float minimum);
}

static void launch_improved_kernel(float *X, float Y, float *max, float *min, float maximum, float minimum,int Inputsize)
 {
    dim3 dimGrid(blocksize,1,1);
    dim3 dimBlock(M,(N-1)/blocksize+1,1);
    sum_improved_kernel<<<dimGrid,dimBlock>>>(float *X, float Y, float *max, float *min, float maximum, float minimum);
}



// Declare error and timing utilities =========================================

#define FATAL(msg, ...) \
    do {\
        fprintf(stderr, "[%s:%d] "msg"\n\n", __FILE__, __LINE__, ##__VA_ARGS__);\
        exit(-1);\
    } while(0)

typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;
void startTime(Timer* timer);
void stopTime(Timer* timer);
float elapsedTime(Timer timer);

// Main function ==============================================================
int main(float *X,float Y, int InputSize) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize GPU/CUDA ----------------------------------------------------

    printf("\nInitializing GPU/CUDA..."); fflush(stdout);
    startTime(&timer);
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Input parameters and host variables -------------------------

    printf("Setting up the problem..."); fflush(stdout);
    startTime(&timer);
     
     unsigned int nSimu, nSteps;
    nSimu = 200;// 200 different predictions for each stock
    nSteps = 100;//100 time points (,time steps)

     float* X;
     int Inputsize=sizeof(float)* N * M;
     int size=sizeof(float);
    cuda_ret = hipMalloc((void**) &X, sizeof(float)*nSimu*nAssets);
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    __syncthreads();

    startTime(&timer);
    launch_simple_kernel(float X, float Y, float max, float min, float maximum, float minimum);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));


    startTime(&timer);
    launch_improved_kernel(float X, float Y, float max, float min, float maximum, float minimum);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    

    stopTime(&timer);
    int InputSize = n *sizeof(float);
    hipMalloc((void**)&X,Inputsize);
    hipMemcpy(X,price,Inputsize, hipMemcpyHostToDevice);

    hipMalloc((void**)&Y,size);
    
    hipMemcpy(Y,Y,size, hipMemcpyDeviceToHost);
    hipFree(X);
    hipFree(Y);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
}
