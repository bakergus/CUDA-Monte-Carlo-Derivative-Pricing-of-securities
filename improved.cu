#include "hip/hip_runtime.h"
project
=======
#define MAX(a,b) ((a) > (b) ? a : b)
#define MIN(a,b) ((a) < (b) ? a : b)
__global__ void sum_improved_kernel(float *X, float Y, float *max, float *min, float maximum, float minimum)
{
__shared__ float partialSum[];

unsigned int t=threadIdx.x;
for(unsigned int stride=blockDim.x;stride>1;stride/=2)
{
__syncthreads();
if(t < stride)
partialSum[t]+=partialSum[t+stride];
max[t]= Max(X[t],X[t+stride]);
min[t]= Min(X[t],X[t+stride]);
}
Y=partialSum[blockDim.x];
maximum = max[blockDim.x];
minimum = min[blockDim.x];
}
